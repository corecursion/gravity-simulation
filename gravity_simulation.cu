#include "hip/hip_runtime.h"
// gravity_simulation.cu

// #include <cstdio>
// __global__ void cuda_hello(){
//     printf("Hello World from GPU!\n");
// }

#include <chrono>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <string>

using namespace std::literals;

#include <glad/glad.h>
#define GLFW_INCLUDE_NONE
#define GLFW_DLL
#include <GLFW/glfw3.h>

#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>

#include "particles.hh"

static void error_callback(int error, const char* description)
{
    fprintf(stderr, "Error: %s\n", description);
}

static void key_callback(GLFWwindow* window, int key, int scancode, int action, int mods)
{
    if (key == GLFW_KEY_ESCAPE && action == GLFW_PRESS)
        glfwSetWindowShouldClose(window, GLFW_TRUE);
}

const char *vertex_shader_text =
    "#version 330 core\n"
    "uniform mat4 model;"
    "uniform mat4 view;"
    "uniform mat4 projection;"
    "layout (location = 0) in vec2 pos;\n"
    "layout (location = 1) in float sz;\n"
    "void main()\n"
    "{\n"
    "    gl_Position = projection * vec4(pos, 0.0, 1.0);\n"
    "    gl_PointSize = sz;\n"
    "}\n";

const char *fragment_shader_text =
    "#version 330 core\n"
    "out vec4 FragColor;\n"
    "void main()\n"
    "{\n"
    "    vec2 coord = gl_PointCoord - vec2(0.5);\n"
    "    if (length(coord) > 0.5) discard;\n"
    "    FragColor = vec4(1.0f, 0.5f, 0.2f, 1.0f);\n"
    "}\n";

unsigned int make_shader_program() {
    // Vertex shader.
    unsigned int vertex_shader = glCreateShader(GL_VERTEX_SHADER);
    glShaderSource(vertex_shader, 1, &vertex_shader_text, NULL);
    glCompileShader(vertex_shader);
    int success;
    char info_log[512];
    glGetShaderiv(vertex_shader, GL_COMPILE_STATUS, &success);
    if (!success) {
        glGetShaderInfoLog(vertex_shader, 512, NULL, info_log);
        throw std::runtime_error("compiling vertex shader failed\n"s + info_log);
    }

    // Fragment shader.
    unsigned int fragment_shader = glCreateShader(GL_FRAGMENT_SHADER);
    glShaderSource(fragment_shader, 1, &fragment_shader_text, NULL);
    glCompileShader(fragment_shader);
    glGetShaderiv(fragment_shader, GL_COMPILE_STATUS, &success);
    if (!success) {
        glGetShaderInfoLog(fragment_shader, 512, NULL, info_log);
        throw std::runtime_error("compiling fragment shader failed\n"s + info_log);
    }

    // Shader program.
    unsigned int shader_program = glCreateProgram();
    glAttachShader(shader_program, vertex_shader);
    glAttachShader(shader_program, fragment_shader);
    glLinkProgram(shader_program);
    glGetProgramiv(shader_program, GL_LINK_STATUS, &success);
    if (!success) {
        glGetProgramInfoLog(shader_program, GL_LINK_STATUS, &success, info_log);
        throw std::runtime_error("linking shaders failed\n"s + info_log);
    }

    return shader_program;
}

const unsigned int SCR_WIDTH = 1500;
const unsigned int SCR_HEIGHT = 1200;

int main2() {
    // cuda_hello<<<1,1>>>();

    GLFWwindow* window;

    glfwSetErrorCallback(error_callback);

    if (!glfwInit())
        exit(EXIT_FAILURE);

    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    window = glfwCreateWindow(SCR_WIDTH, SCR_HEIGHT, "Gravity Simulation", NULL, NULL);
    if (!window) {
        glfwTerminate();
        exit(EXIT_FAILURE);
    }


    glfwSetKeyCallback(window, key_callback);

    glfwMakeContextCurrent(window);

    if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
        throw std::runtime_error("Failed to initialize GLAD");

    glfwSwapInterval(1);

    unsigned int shader_program = make_shader_program();
    glUseProgram(shader_program);

    glEnable(GL_PROGRAM_POINT_SIZE);
    glEnable(GL_BLEND);
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);

    // Matrixes
    glm::mat4 projection = glm::mat4(1.0f);
    glm::mat4 view = glm::mat4(1.0f);
    // projection = glm::perspective(glm::radians(45.0f), (float)SCR_WIDTH / (float)SCR_HEIGHT, 0.1f, 100.0f);
    projection = glm::ortho(0.0F, (float)SCR_WIDTH, 0.0F, (float)SCR_HEIGHT, -1.0F, +1.0F);
    // view       = glm::translate(view, glm::vec3(0.0f, 0.0f, -3.0f));
    glUniformMatrix4fv(glGetUniformLocation(shader_program, "projection"), 1, GL_FALSE, &projection[0][0]);
    glUniformMatrix4fv(glGetUniformLocation(shader_program, "view"), 1, GL_FALSE, &view[0][0]);

    Particles particles = init_particle_grid(SCR_WIDTH, SCR_HEIGHT, /*radius=*/250, /*max_velocity=*/10, /*step=*/14);
    std::cout << particles.size() << " particles" << std::endl;
    bool flip = false;

    auto ts1 = std::chrono::system_clock::now();
    auto ts2 = ts1;
    while (!glfwWindowShouldClose(window))
    {
        // float ratio;
        int width, height;

        glfwGetFramebufferSize(window, &width, &height);

        glViewport(0, 0, width, height);

        glm::mat4 model = glm::mat4(1.0f);
        glUniformMatrix4fv(glGetUniformLocation(shader_program, "model"), 1, GL_FALSE, &model[0][0]);

        ts2 = std::chrono::system_clock::now();
        double delta = std::chrono::duration<double>(ts2-ts1).count();
        if (delta == 0.0) throw std::runtime_error("zero time passed");
        if (delta > 0.5) {
            std::cout << std::fixed << delta << "s hitch" << std::endl;
            delta = 0.5;
        }
        accelerate_particles(particles, delta, flip);
        move_particles(particles, delta, flip);
        draw_particles(particles, shader_program);

        glfwSwapBuffers(window);
        glfwPollEvents();

        flip = !flip;
        ts1 = std::move(ts2);
    }

    glfwTerminate();
    return 0;
}

int main() {
    try {
        main2();
    } catch(const std::exception& err) {
        std::cout << "EXCEPTION: " << err.what() << std::endl;
    } catch(...) {
        std::cout << "UNKNOWN EXCEPTION" << std::endl;
    }
}
