#include "hip/hip_runtime.h"
// gravity_simulation.cu

// #include <cstdio>
// __global__ void cuda_hello(){
//     printf("Hello World from GPU!\n");
// }

#include <chrono>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <string>

using namespace std::literals;

#include <glad/glad.h>
#define GLFW_INCLUDE_NONE
#define GLFW_DLL
#include <GLFW/glfw3.h>

#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>

#include "particles.hh"

static void error_callback(int error, const char* description)
{
    fprintf(stderr, "Error: %s\n", description);
}

static void key_callback(GLFWwindow* window, int key, int scancode, int action, int mods)
{
    if (key == GLFW_KEY_ESCAPE && action == GLFW_PRESS)
        glfwSetWindowShouldClose(window, GLFW_TRUE);
}

const char *vertex_shader_text =
    "#version 330 core\n"
    "uniform mat4 model;"
    "uniform mat4 view;"
    "uniform mat4 projection;"
    "layout (location = 0) in vec2 pos;\n"
    "layout (location = 1) in float sz;\n"
    "void main()\n"
    "{\n"
    "   gl_Position = projection * vec4(pos, 0.0, 1.0);\n"
    "   gl_PointSize = sz;\n"
    "}\n";

const char *fragment_shader_text =
    "#version 330 core\n"
    "out vec4 FragColor;\n"
    "void main()\n"
    "{\n"
    "    FragColor = vec4(1.0f, 0.5f, 0.2f, 1.0f);\n"
    "}\n";

unsigned int make_shader_program() {
    // Vertex shader.
    unsigned int vertex_shader = glCreateShader(GL_VERTEX_SHADER);
    glShaderSource(vertex_shader, 1, &vertex_shader_text, NULL);
    glCompileShader(vertex_shader);
    int success;
    char info_log[512];
    glGetShaderiv(vertex_shader, GL_COMPILE_STATUS, &success);
    if (!success) {
        glGetShaderInfoLog(vertex_shader, 512, NULL, info_log);
        throw std::runtime_error("compiling vertex shader failed\n"s + info_log);
    }

    // Fragment shader.
    unsigned int fragment_shader = glCreateShader(GL_FRAGMENT_SHADER);
    glShaderSource(fragment_shader, 1, &fragment_shader_text, NULL);
    glCompileShader(fragment_shader);
    glGetShaderiv(fragment_shader, GL_COMPILE_STATUS, &success);
    if (!success) {
        glGetShaderInfoLog(fragment_shader, 512, NULL, info_log);
        throw std::runtime_error("compiling fragment shader failed\n"s + info_log);
    }

    // Shader program.
    unsigned int shader_program = glCreateProgram();
    glAttachShader(shader_program, vertex_shader);
    glAttachShader(shader_program, fragment_shader);
    glLinkProgram(shader_program);
    glGetProgramiv(shader_program, GL_LINK_STATUS, &success);
    if (!success) {
        glGetProgramInfoLog(shader_program, GL_LINK_STATUS, &success, info_log);
        throw std::runtime_error("linking shaders failed\n"s + info_log);
    }

    return shader_program;
}

const unsigned int SCR_WIDTH = 800;
const unsigned int SCR_HEIGHT = 800;

int main2() {
    // cuda_hello<<<1,1>>>();

    GLFWwindow* window;

    glfwSetErrorCallback(error_callback);

    if (!glfwInit())
        exit(EXIT_FAILURE);

    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    window = glfwCreateWindow(SCR_WIDTH, SCR_HEIGHT, "Gravity Simulation", NULL, NULL);
    if (!window) {
        glfwTerminate();
        exit(EXIT_FAILURE);
    }


    glfwSetKeyCallback(window, key_callback);

    glfwMakeContextCurrent(window);

    if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
        throw std::runtime_error("Failed to initialize GLAD");
    // gladLoadGL();



    glfwSwapInterval(1);


    unsigned int shader_program = make_shader_program();
    glUseProgram(shader_program);

    glEnable(GL_PROGRAM_POINT_SIZE);
 

    // Matrixes
    glm::mat4 projection = glm::mat4(1.0f);
    glm::mat4 view = glm::mat4(1.0f);
    // projection = glm::perspective(glm::radians(45.0f), (float)SCR_WIDTH / (float)SCR_HEIGHT, 0.1f, 100.0f);
    projection = glm::ortho(0.0F, (float)SCR_WIDTH, 0.0F, (float)SCR_HEIGHT, -1.0F, +1.0F);
    // view       = glm::translate(view, glm::vec3(0.0f, 0.0f, -3.0f));
    glUniformMatrix4fv(glGetUniformLocation(shader_program, "projection"), 1, GL_FALSE, &projection[0][0]);
    glUniformMatrix4fv(glGetUniformLocation(shader_program, "view"), 1, GL_FALSE, &view[0][0]);


    Particles particles = init_particle_grid(SCR_WIDTH, SCR_HEIGHT, /*max_velocity=*/10, /*step=*/25);
    std::cout << particles.size() << " particles" << std::endl;
    bool flip = false;

    auto ts1 = std::chrono::system_clock::now();
    auto ts2 = ts1;
    // size_t count = 0;
    while (!glfwWindowShouldClose(window))
    {
        // float ratio;
        int width, height;

        glfwGetFramebufferSize(window, &width, &height);
        // ratio = width / (float) height;

        // glMatrixMode(GL_PROJECTION);
        // glLoadIdentity();
        glViewport(0, 0, width, height);
        // glMatrixMode(GL_MODELVIEW);
        // glLoadIdentity();
        // glOrtho(0, width-1, 0, height-1, -1, 1);
        // glClear(GL_COLOR_BUFFER_BIT);

        glm::mat4 model = glm::mat4(1.0f);
        glUniformMatrix4fv(glGetUniformLocation(shader_program, "model"), 1, GL_FALSE, &model[0][0]);

        ts2 = std::chrono::system_clock::now();
        double delta = std::chrono::duration<double>(ts2-ts1).count();
        if (delta == 0.0) throw std::runtime_error("zero time passed");
        if (delta > 0.5) {
            std::cout << std::fixed << delta << "s hitch" << std::endl;
            delta = 0.5;
        }
        accelerate_particles(particles, delta, flip);
        move_particles(particles, delta, flip);
        draw_particles(particles, shader_program);

        glfwSwapBuffers(window);
        glfwPollEvents();

        flip = !flip;
        ts1 = std::move(ts2);
        // if (++count == 3) exit(0);
    }

    glfwTerminate();
    return 0;
}

int main() {
    try {
        main2();
    } catch(const std::exception& err) {
        std::cout << "EXCEPTION: " << err.what() << std::endl;
    } catch(...) {
        std::cout << "UNKNOWN EXCEPTION" << std::endl;
    }
}



// #include <glad/glad.h>
// #include <GLFW/glfw3.h>

// #include <iostream>

// void framebuffer_size_callback(GLFWwindow* window, int width, int height);
// void processInput(GLFWwindow *window);

// // settings
// const unsigned int SCR_WIDTH = 800;
// const unsigned int SCR_HEIGHT = 600;

// const char *vertexShaderSource = "#version 330 core\n"
//     "layout (location = 0) in vec3 aPos;\n"
//     "void main()\n"
//     "{\n"
//     "   gl_Position = vec4(aPos.x, aPos.y, aPos.z, 1.0);\n"
//     "}\0";
// const char *fragmentShaderSource = "#version 330 core\n"
//     "out vec4 FragColor;\n"
//     "void main()\n"
//     "{\n"
//     "   FragColor = vec4(1.0f, 0.5f, 0.2f, 1.0f);\n"
//     "}\n\0";

// int main()
// {
//     // glfw: initialize and configure
//     // ------------------------------
//     glfwInit();
//     glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
//     glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
//     glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

// #ifdef __APPLE__
//     glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
// #endif

//     // glfw window creation
//     // --------------------
//     GLFWwindow* window = glfwCreateWindow(SCR_WIDTH, SCR_HEIGHT, "LearnOpenGL", NULL, NULL);
//     if (window == NULL)
//     {
//         std::cout << "Failed to create GLFW window" << std::endl;
//         glfwTerminate();
//         return -1;
//     }
//     glfwMakeContextCurrent(window);
//     glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);

//     // glad: load all OpenGL function pointers
//     // ---------------------------------------
//     if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
//     {
//         std::cout << "Failed to initialize GLAD" << std::endl;
//         return -1;
//     }


//     // build and compile our shader program
//     // ------------------------------------
//     // vertex shader
//     unsigned int vertexShader = glCreateShader(GL_VERTEX_SHADER);
//     glShaderSource(vertexShader, 1, &vertexShaderSource, NULL);
//     glCompileShader(vertexShader);
//     // check for shader compile errors
//     int success;
//     char infoLog[512];
//     glGetShaderiv(vertexShader, GL_COMPILE_STATUS, &success);
//     if (!success)
//     {
//         glGetShaderInfoLog(vertexShader, 512, NULL, infoLog);
//         std::cout << "ERROR::SHADER::VERTEX::COMPILATION_FAILED\n" << infoLog << std::endl;
//     }
//     // fragment shader
//     unsigned int fragmentShader = glCreateShader(GL_FRAGMENT_SHADER);
//     glShaderSource(fragmentShader, 1, &fragmentShaderSource, NULL);
//     glCompileShader(fragmentShader);
//     // check for shader compile errors
//     glGetShaderiv(fragmentShader, GL_COMPILE_STATUS, &success);
//     if (!success)
//     {
//         glGetShaderInfoLog(fragmentShader, 512, NULL, infoLog);
//         std::cout << "ERROR::SHADER::FRAGMENT::COMPILATION_FAILED\n" << infoLog << std::endl;
//     }
//     // link shaders
//     unsigned int shaderProgram = glCreateProgram();
//     glAttachShader(shaderProgram, vertexShader);
//     glAttachShader(shaderProgram, fragmentShader);
//     glLinkProgram(shaderProgram);
//     // check for linking errors
//     glGetProgramiv(shaderProgram, GL_LINK_STATUS, &success);
//     if (!success) {
//         glGetProgramInfoLog(shaderProgram, 512, NULL, infoLog);
//         std::cout << "ERROR::SHADER::PROGRAM::LINKING_FAILED\n" << infoLog << std::endl;
//     }
//     glDeleteShader(vertexShader);
//     glDeleteShader(fragmentShader);

//     // set up vertex data (and buffer(s)) and configure vertex attributes
//     // ------------------------------------------------------------------
//     float vertices[] = {
//         -0.5f, -0.5f, 0.0f, // left  
//          0.5f, -0.5f, 0.0f, // right 
//          0.0f,  0.5f, 0.0f  // top   
//     }; 

//     unsigned int VBO, VAO;
//     glGenVertexArrays(1, &VAO);
//     glGenBuffers(1, &VBO);
//     // bind the Vertex Array Object first, then bind and set vertex buffer(s), and then configure vertex attributes(s).
//     glBindVertexArray(VAO);

//     glBindBuffer(GL_ARRAY_BUFFER, VBO);
//     glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);

//     glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 3 * sizeof(float), (void*)0);
//     glEnableVertexAttribArray(0);

//     // note that this is allowed, the call to glVertexAttribPointer registered VBO as the vertex attribute's bound vertex buffer object so afterwards we can safely unbind
//     glBindBuffer(GL_ARRAY_BUFFER, 0); 

//     // You can unbind the VAO afterwards so other VAO calls won't accidentally modify this VAO, but this rarely happens. Modifying other
//     // VAOs requires a call to glBindVertexArray anyways so we generally don't unbind VAOs (nor VBOs) when it's not directly necessary.
//     glBindVertexArray(0); 


//     // uncomment this call to draw in wireframe polygons.
//     //glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);

//     // render loop
//     // -----------
//     while (!glfwWindowShouldClose(window))
//     {
//         // input
//         // -----
//         processInput(window);

//         // render
//         // ------
//         glClearColor(0.2f, 0.3f, 0.3f, 1.0f);
//         glClear(GL_COLOR_BUFFER_BIT);

//         // draw our first triangle
//         glUseProgram(shaderProgram);
//         glBindVertexArray(VAO); // seeing as we only have a single VAO there's no need to bind it every time, but we'll do so to keep things a bit more organized
//         // glDrawArrays(GL_POINTS, 0, 3);
//         // glDrawArrays(GL_LINE_LOOP, 0, 3);
//         glDrawArrays(GL_TRIANGLES, 0, 3);
//         // glBindVertexArray(0); // no need to unbind it every time 
 
//         // glfw: swap buffers and poll IO events (keys pressed/released, mouse moved etc.)
//         // -------------------------------------------------------------------------------
//         glfwSwapBuffers(window);
//         glfwPollEvents();
//     }

//     // optional: de-allocate all resources once they've outlived their purpose:
//     // ------------------------------------------------------------------------
//     glDeleteVertexArrays(1, &VAO);
//     glDeleteBuffers(1, &VBO);
//     glDeleteProgram(shaderProgram);

//     // glfw: terminate, clearing all previously allocated GLFW resources.
//     // ------------------------------------------------------------------
//     glfwTerminate();
//     return 0;
// }

// // process all input: query GLFW whether relevant keys are pressed/released this frame and react accordingly
// // ---------------------------------------------------------------------------------------------------------
// void processInput(GLFWwindow *window)
// {
//     if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
//         glfwSetWindowShouldClose(window, true);
// }

// // glfw: whenever the window size changed (by OS or user resize) this callback function executes
// // ---------------------------------------------------------------------------------------------
// void framebuffer_size_callback(GLFWwindow* window, int width, int height)
// {
//     // make sure the viewport matches the new window dimensions; note that width and 
//     // height will be significantly larger than specified on retina displays.
//     glViewport(0, 0, width, height);
// }