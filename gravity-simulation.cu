// gravity_simulation.cu
// Copyright (C) 2023 by Shawn Yarbrough

#include <chrono>
#include <cstdlib>
#include <cstdio>
#include <fstream>
#include <iostream>
#include <iterator>
#include <string>

using namespace std::literals;

#include "graphics.hh"
#include "particles.hh"
#include "utility.hh"

const unsigned int SCR_WIDTH = 1920;
const unsigned int SCR_HEIGHT = 1080;

Particles load_particles_from_csv(const std::string& csv_filename) {
    Particles particles;

    Csv::Parser csv;
    std::vector<std::vector<Csv::CellReference>> cells;
    std::ifstream ifile(csv_filename, std::ios::binary);
    std::string data((std::istreambuf_iterator<char>(ifile)), (std::istreambuf_iterator<char>()));
    csv.parseTo(data, cells);

    if (cells.size() > 0) {
        std::vector<std::string> headings;
        size_t cols = cells.size();
        size_t rows = cells[0].size();
        for (std::size_t col = 0; col < cols; ++col) {
            if (cells[col].size() != rows)
                throw std::runtime_error(".csv column #"+std::to_string(col+1)+" unexpected size");
            const auto& cell = cells[col][0];
            if (cell.getType() != Csv::CellType::String)
                throw std::runtime_error("unexpected type for string heading column #"+std::to_string(col+1));
            std::optional<std::string> s = cell.getCleanString().value();
            headings.push_back(utility::strip(s.value_or("")));
        }
        size_t next_id = 0;
        for (std::size_t row = 1; row < rows; ++row) {
            Particle p;
            p.id = next_id++;
            for (std::size_t col = 0; col < cols; ++col) {
                const auto& cell = cells[col][row];
                if (cell.getType() != Csv::CellType::Double)
                    throw std::runtime_error("unexpected type for number in column #"+std::to_string(col+1)+" row #"+std::to_string(row+1));
                double d = cell.getDouble().value();
                const std::string& heading = headings[col];
                if (heading == "xposition")
                    p.position[0] = d;
                else if (heading == "yposition")
                    p.position[1] = d;
                else if (heading == "xvelocity")
                    p.velocity[0] = d;
                else if (heading == "yvelocity")
                    p.velocity[1] = d;
                else if (heading == "diameter")
                    p.diameter = d;
                else
                    throw std::runtime_error("unexpected name for .csv col #"+std::to_string(col+1)+": "+heading);
            }
            particles.push_back(std::move(p));
        }
    }

    return particles;
}

int main2(int argc, char* argv[]) {
    auto [window, shader_program] = graphics::setup_app_window(SCR_WIDTH, SCR_HEIGHT);

    Particles particles;
    if (argc > 1) {
        particles = load_particles_from_csv(argv[1]);
    } else {
        particles = Particle::init_particle_grid(SCR_WIDTH, SCR_HEIGHT, /*radius=*/1000, /*max_velocity=*/10, /*step=*/20);
    }
    std::cout << particles.size() << " particles" << std::endl;

    auto ts1 = std::chrono::system_clock::now();
    auto ts2 = ts1;
    while (!glfwWindowShouldClose(window))
    {
        graphics::center_app_window(window, shader_program);

        ts2 = std::chrono::system_clock::now();
        double delta = std::chrono::duration<double>(ts2-ts1).count();
        if (delta == 0.0) throw std::runtime_error("zero time passed");
        if (delta > 0.2) {
            std::cout << std::fixed << delta << "s hitch" << std::endl;
            delta = 0.2;
        }
        particles = Particle::accelerate_particles(particles, delta);
        Particle::move_particles(particles, delta);
        Particle::draw_particles(particles, shader_program);

        glfwSwapBuffers(window);
        glfwPollEvents();

        ts1 = std::move(ts2);
    }

    glfwTerminate();
    return EXIT_SUCCESS;
}

int main(int argc, char* argv[]) {
    try {
        return main2(argc, argv);
    } catch(const std::exception& err) {
        std::cout << "EXCEPTION: " << err.what() << std::endl;
        return 1;
    } catch(...) {
        std::cout << "UNKNOWN EXCEPTION" << std::endl;
        return 2;
    }
}
